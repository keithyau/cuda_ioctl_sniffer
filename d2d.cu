#include <stdio.h>
#include <hip/hip_runtime.h>
// dmesg -w | grep -Ei "p2p|fault"

int main(int argc, char *argv[]) {
  printf("***** malloc 1\n");
  hipSetDevice(1);
  float *b = NULL;
  hipMalloc(&b, 0x10000);

  printf("***** malloc 0\n");
  hipSetDevice(0);
  float *a = NULL;
  hipMalloc(&a, 0x10000);
  hipSetDevice(1);

  printf("***** enable p2p\n");
  hipDeviceEnablePeerAccess(0, 0);

  printf("***** hipMemcpyDtoD %p %p\n", a, b);
  hipMemcpyDtoD((hipDeviceptr_t)a, (hipDeviceptr_t)b, 0x1000);

  printf("***** done\n");
}